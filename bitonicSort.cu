// test codes for sorting 64M float keys
// only kernel codes. uses 1 float array to sort in-place.
// uses dynamic parallelism feature of cuda
// array size needs to be integer power of 2
// arary size needs to be at least 8192
// benchmark data:

/*
Array elements  GT1030		    std::sort 	        GTX1080ti 
               (benchmark)   (1 core )             (guesstimate)
               (no overclock)
1024            not applicable                            -
2048		    not applicable				      -
4096		    not applicable				      -
8192		    363	    µs		  114 µs		      -
16k			    463	    ms		  248 µs		      -
32k			    746 	µs		  536	µs		      -
64k			    1.23	ms		  1.15ms		      -
128k		    2.32	ms		  2.46ms		      -
256k		    4.87	ms		  5.4	ms		~1.5+ 0.3	ms
512k		    8.72	ms		  11.7ms		~3	+ 0.5	ms
1M			    18.3	ms		  22	ms		~6  + 1.2	ms
2M			    39      ms		  48	ms		~12 + 2.7	ms
4M			    86      ms		  101 ms		~23 + 6.3	ms
8M			    187	    ms		  211 ms		~47 + 14	ms
16M			    407	    ms		  451 ms		~95 + 32	ms
32M			    883	    ms		  940	ms		~190+ 70	ms
64M			    1.93	s		  2.0 s		    ~380+ 150	ms
(float keys)    (copy+kernel )			(copy + kernel)
                                        (using same pcie)
pcie v2.0 4x: 1.4GB/s
fx8150 @ 3.6GHz
4GB RAM 1333MHz
(single channel DDR3)
*/

static constexpr int pow(int base, int exp)
{     
     int res = base;
     int exp_ = exp;
     
     while (exp_ > 1)
     {          
          exp_--;
          res = res * base;
     }

     return res;    
}


constexpr int n = pow(2, 25); // 64M elements
constexpr int l2n= 25;  // log2(n)


// shared memory per block, also number of work per block (2048=minimum, 4096=moderate, 8192=maximum).
constexpr int sharedSize= 8192; 
constexpr int l22k= 13; // log2(sharedSize)
__device__ void compareSwap(float & var1, float &var2, bool dir)
{
     if(var1>var2 && dir)
     {                
               float tmp = var1;
               var1=var2;
               var2=tmp;
     }
     else if(var1<var2 && !dir)
     {
               float tmp = var1;
               var1=var2;
               var2=tmp;   
     }
}
__global__ void computeBox(float * __restrict__ data, const int boxSize, const int leapSize)
{
     const int index = (threadIdx.x + blockIdx.x*blockDim.x);
     const bool dir = ((index%boxSize)<(boxSize/2));
     const int indexOffset = (index / leapSize)*leapSize;
     
     compareSwap(data[index+indexOffset],data[index+indexOffset+leapSize],dir);
}
__global__ void computeBoxForward(float * __restrict__ data, const int boxSize, const int leapSize)
{
     const int index = (threadIdx.x + blockIdx.x*blockDim.x);
     const bool dir = true;
     const int indexOffset = (index / leapSize)*leapSize;
     
     compareSwap(data[index+indexOffset],data[index+indexOffset+leapSize],dir);
}
__device__ void computeBoxShared(float * __restrict__ data, const int boxSize, const int leapSize, const int work)
{
     const int index = threadIdx.x+work*1024;
     const bool dir = ((index%boxSize)<(boxSize/2));
     const int indexOffset = (index / leapSize)*leapSize;
     
     compareSwap(data[index+indexOffset],data[index+indexOffset+leapSize],dir);
}
__device__ void computeBoxForwardShared(float * __restrict__ data, const int boxSize, const int leapSize, const int work)
{
     const int index = threadIdx.x + work*1024;
     const bool dir = true;
     const int indexOffset = (index / leapSize)*leapSize;
     
     compareSwap(data[index+indexOffset],data[index+indexOffset+leapSize],dir);
}
__global__ void bitonicSharedSort(float * __restrict__ data)
{
     const int offset = blockIdx.x * sharedSize;
     __shared__ float sm[sharedSize];
     const int nCopy = sharedSize / 1024;
     const int nWork = sharedSize / 2048;
     for(int i=0;i<nCopy;i++)
     {
          sm[threadIdx.x+i*1024]      = data[threadIdx.x+offset+i*1024];
     }
     __syncthreads();
     int boxSize = 2;
     for(int i=0;i<l22k-1;i++)
     {                       
          for(int leapSize = boxSize/2;leapSize>0;leapSize /= 2)
          {                             
               for(int work=0;work<nWork;work++)
               {                  
                    computeBoxShared(sm,boxSize,leapSize,work);
               }                          
               __syncthreads();
          }
          boxSize*=2;
     }
     
     for(int leapSize = boxSize/2;leapSize>0;leapSize /= 2)
     {           
          for(int work=0;work<nWork;work++)
          {         
               computeBoxForwardShared(sm,boxSize,leapSize,work);
          }                 
          __syncthreads();     
     }
          
     for(int i=0;i<nCopy;i++)
     {
          data[threadIdx.x+offset+i*1024] = sm[threadIdx.x+i*1024];               		      
     }
}
__global__ void bitonicSharedMergeLeaps(float * __restrict__ data, const int boxSizeP, const int leapSizeP)
{
     const int offset = blockIdx.x * sharedSize;
     __shared__ float sm[sharedSize];
     const int nCopy = sharedSize / 1024;
     const int nWork = sharedSize / 2048;
     for(int i=0;i<nCopy;i++)
     {
          sm[threadIdx.x+i*1024] = data[threadIdx.x+offset+i*1024];		 
     }
     __syncthreads();
     
     for(int leapSize = leapSizeP;leapSize>0;leapSize /= 2)
     {                                               
               for(int work=0;work<nWork;work++)
               {
               const int index = threadIdx.x+work*1024;
               const int index2 = threadIdx.x+work*1024+blockIdx.x*blockDim.x*nWork;
               const bool dir = ((index2%boxSizeP)<(boxSizeP/2));
               const int indexOffset = (index / leapSize)*leapSize;
               
               compareSwap(sm[index+indexOffset],sm[index+indexOffset+leapSize],dir);
               }                          
          __syncthreads();
     }

     for(int i=0;i<nCopy;i++)
     {
     data[threadIdx.x+offset+i*1024] = sm[threadIdx.x+i*1024];               		 	    
     }
}

// launch this with 1 cuda thread
// dynamic parallelism = needs something newer than cc v3.0
//extern "C"
//__global__ 
void bitonicSort(float * __restrict__ data)
{     

     bitonicSharedSort<<<(n/sharedSize),1024>>>(data);
     hipDeviceSynchronize();       

     int boxSize = sharedSize;
     for(int i=l22k-1;i<l2n-1;i++)
     {
              if(boxSize>sharedSize)
              {
                   int leapSize= boxSize/2;
                   for(;leapSize>sharedSize/2;leapSize /= 2)
                   {                                               
                        computeBox<<<(n/1024)/2,1024>>>(data,boxSize,leapSize);    
                        //cudaDeviceSynchronize();                                              											  
                   }
                   hipDeviceSynchronize();
                   bitonicSharedMergeLeaps<<<(n/sharedSize),1024>>>(data,boxSize, leapSize);
                   hipDeviceSynchronize();
              }
              else
              {
                   bitonicSharedMergeLeaps<<<(n/sharedSize),1024>>>(data,boxSize, sharedSize/2);
                   hipDeviceSynchronize();
              }
         boxSize*=2;
         hipDeviceSynchronize();
     }
     
     
     for(int leapSize = boxSize/2;leapSize>0;leapSize /= 2)
     {                    
         computeBoxForward<<<(n/1024)/2,1024>>>(data,boxSize,leapSize); 
     }
     
     hipDeviceSynchronize();          		  
}	

void bitonicSortNoShared(float * __restrict__ data)
{            
     int boxSize = 2;
     for(int i=0;i<l2n-1;i++)
     {
          for(int leapSize = boxSize/2;leapSize>0;leapSize /= 2)
          {               
               computeBox<<<(n/1024)/2,1024>>>(data,boxSize,leapSize);
          }
          hipDeviceSynchronize();
          boxSize*=2;
     }    

     for(int leapSize = boxSize/2;leapSize>0;leapSize /= 2)
     {        
          computeBoxForward<<<(n/1024)/2,1024>>>(data,boxSize,leapSize);
     }
     hipDeviceSynchronize();
}	

#include <vector>
#include <random>
#include <iostream>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <chrono>
#include <stdio.h>
#include <execution>

void TestHostQuickSort(std::vector<float>& a, std::vector<float>& b)
{

     //std::sort(a.begin(), a.end());

     std::sort(std::execution::par_unseq, a.begin(), a.end());     

     //  for (int i = 0; i < 5; i++)
     // {
     //     std::cout << a[n - 1 - i] << std::endl;
     // }

}

void TestDeviceBitonicSort(std::vector<float>& a)
{
     hipError_t err = hipGetLastError();

     float * d_a = nullptr;

     hipMalloc((void**)&d_a, sizeof(float) * n);
     hipMemcpy(d_a, a.data(), sizeof(float) * n, hipMemcpyDefault);
     //bitonicSort<<<1, 1>>>(d_a);
     //bitonicSort(d_a);
     bitonicSortNoShared(d_a);
     hipDeviceSynchronize();

     hipMemcpy(a.data(), d_a, sizeof(float) * n, hipMemcpyDefault);     

     err = hipGetLastError();
     std::cout << hipGetErrorString(err) << std::endl;

     //for (int i = 0; i < 5; i++)
     //{
//
     //    std::cout << a[n - 1 - i] << std::endl;
//
     //}
}

void ValidateResult(std::vector<float>& a)
{

     for (int i = 0; i < n - 1; i++){

          if (a[i] > a[i + 1]) { 

               std::cout << "same or less on " << i << ", " << a[i] << " " << a[i + 1] << std::endl;

          }

     }

}

int main()
{



     std::mt19937 mtRand(2023);
     std::uniform_int_distribution<int> dist1(-n, n);
     std::chrono::system_clock::time_point start;
     std::chrono::microseconds us;

     std::cout << "array size: " << n << std::endl;
    
     std::cout << std::fixed; 

     std::vector<float> a;
     //std::vector<float> b;
     //std::vector<float> c;

     a.resize(n);
     //b.resize(n);
     //c.resize(n);

     for (int i = 0; i < n; i++)
     {
          a[i] = dist1(mtRand);
          //b[i] = a[i];
     }

     //FILE* filePtr;
//
     //filePtr = fopen("unsorted.txt", "w+");
     //for (int i = 0; i < n; i+=50)
     //{
     //     fprintf(filePtr, "%d %.2f \n", i, a[i]);
     //}
     //fclose(filePtr);

     // start = std::chrono::system_clock::now();
     // TestHostQuickSort(a, b);
     // us = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now() - start);
     // std::cout << "host took " << us.count() << "us" << std::endl;

     // for (int i = 0; i < n; i++)
     // {
     //     c[i] = a[i];
     // }     

     // for (int i = 0; i < n; i++)
     // {
     //     a[i] = b[i];
     // }     


     start = std::chrono::system_clock::now();
     TestDeviceBitonicSort(a);
     
     us = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now() - start);
     std::cout << "device took " << us.count() << "us" << std::endl;


     //FILE* filePtr;

     //filePtr = fopen("res.txt", "w+");
     //for (int i = 0; i < n; i+=50)
     //{
     //     fprintf(filePtr, "%d %.2f \n", i, a[i]);
     //}
     //fclose(filePtr);


     //validate

     // int sum = 0;

     // for (int i = 0; i < n; i++)
     // {          
     //      sum += abs(c[i] - a[i]);
     // }
     // printf("%d \n", sum);





}


